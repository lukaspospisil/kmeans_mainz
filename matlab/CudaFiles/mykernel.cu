
#include <hip/hip_runtime.h>

__global__ void mykernel( double *Gamma, double *g, double *Theta, double *X, int N, int K, int T) {

	/* compute kernel index */
	int t = blockIdx.x*blockDim.x + threadIdx.x;

	if(t<T){
		int mink;
		
		/* compute g(:,t) */
		for(int k=0;k<K;k++){
			/* compute dot product g(k,t) = <X(:,t) - Theta(:,k),X(:,t) - Theta(:,k)> */
			g[t*K+k] = 0;
			for(int n=0;n<N;n++){
				g[t*K+k] += (X[t*N+n] - Theta[k*N+n])*(X[t*N+n] - Theta[k*N+n]);
			}
			
			/* if this is first row, then Gamma(k,t) is minimal value */
			if(k==0){
				mink=0; /* index k with min value of g(:,t) */
				Gamma[t*K+k] = 1;
			} else {
				/* is this smaller value then previous one? */
				if(g[t*K+k] < g[t*K+mink]){
					/* old one is not min, set it equal to zero */
					Gamma[t*K+mink] = 0;
					mink=k;
					Gamma[t*K+k] = 1;
				} else {
					/* it is not min */
					Gamma[t*K+k] = 0;
				}
			}
		}

	}
}
